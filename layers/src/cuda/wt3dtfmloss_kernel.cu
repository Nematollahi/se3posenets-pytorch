#ifdef __cplusplus
extern "C" {
#endif


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <assert.h>

#define WARPSIZE 32

__constant__ float constTfms[15000];  // ... or some other big enough number

// Warp-shuffle to compute the sum across the warp very efficiently
__inline__ __device__
float warpReduceSum(float val) {
  for (int offset = WARPSIZE/2; offset > 0; offset /= 2)
    val += __shfl_down(val, offset);
  return val;
}

/// Get the (batch,row,col) indices corresponding to a given thread index (3D point index)
__device__ void getCoordinates_2(const int tid, const int nrows, const int ncols,
                                 int &batch, int &row, int &col)
{
    // Get col id
    int id = tid;
    col = id % ncols;
    id = id / ncols;

    // Get row id
    row = id % nrows;
    id = id / nrows;

    // Get batch id
    batch = id;
}

// Sign of a number
__inline__ __device__
int sgn(float val) {
    return (float(0) < val) - (val < float(0));
}

// =============== FWD PASS ================== //

///////////// Kernel
// Compute the loss by transforming each input point by all the "k" transforms, measuring the error
// between the prediction and the target and weighing the corresponding error by the predicted mask weight
__global__ void computeLoss(const float *inputpts, const float *masks, const float *targetpts,
                            float *devLoss, int nrows, int ncols, int npoints, int nSE3,
                            int ps0, int ps1, int ps2, int ps3,
                            int ms0, int ms1, int ms2, int ms3,
                            int ts0, int ts1, int ts2, int ts3)
{
    // Get the index of the point
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Since they are 1D only

    // Create a shared memory buffer for storing the gradients w.r.t a single transform
    extern __shared__ float sharedLoss[];

    // Declare temp vars
    int tid = threadIdx.x; // Id of thread in local block
    int nThreads = blockDim.x;

    // Compute loss only if the point is within limits
    sharedLoss[tid] = 0; // Initialize to zero
    if (id < npoints)
    {
        // Get the batch, row and column indices
        int b,r,c;
        getCoordinates_2(id, nrows, ncols, b, r, c);

        // Get 3D input point (p)
        int valp = b*ps0 + r*ps2 + c*ps3; // Don't add stride along 3D dim
        float x = *(inputpts + 0*ps1 + valp);
        float y = *(inputpts + 1*ps1 + valp);
        float z = *(inputpts + 2*ps1 + valp);

        // Get 3D target point (pt)
        float xt = *(targetpts + 0*ps1 + valp);
        float yt = *(targetpts + 1*ps1 + valp);
        float zt = *(targetpts + 2*ps1 + valp);

        // Compute sum_k w_k * ||R_k*p + t_k - pt||^2 across the different SE3s
        int valm = b*ms0 + r*ms2 + c*ms3;
        for (int k = 0; k < nSE3; k++)
        {
            // Compute transformed 3D point: p' = (R_k*p + t_k) (for X,Y,Z coordinates)
            float *T = constTfms + b*ts0 + k*ts1;   // Get the 'k'th transform
            float xp = (T[0] * x + T[1] * y + T[2]  * z + T[3]);  // (R_k * p_x + t_k)
            float yp = (T[4] * x + T[5] * y + T[6]  * z + T[7]);  // (R_k * p_y + t_k)
            float zp = (T[8] * x + T[9] * y + T[10] * z + T[11]); // (R_k * p_z + t_k)

            // Compute 3D squared-error between target pts & predicted points (powf is 2x slower)
            float err = (xp-xt)*(xp-xt) + (yp-yt)*(yp-yt) + (zp-zt)*(zp-zt);

            // Weight the error by the mask weight
            float w_k = *(masks + k*ms1 + valm); // Get the weight for the 'k'th component of the error

            // Store scaled loss in shared memory
            sharedLoss[tid] += w_k * err;
        }
    }
    __syncthreads();

    // === Do the parallel reduce for that particular transform dimension
    // === ASSUMPTION: We have power of 2 block sizes!
    // From: Slide 22 of http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
    for(unsigned int s = nThreads/2; s>=32; s>>=1)
    {
        // Second nThreads/2 elements will be added to first nThreads/2 elements, then
        // Second nThreads/4 elements will be added to first nThreads/4 elements and so on!
        if (tid < s)
            sharedLoss[tid] += sharedLoss[tid + s];
        __syncthreads();
    }

    // This uses warp-shuffle to compute the sum across a warp (32 threads)
    // Note that for this to work, you have to have run the loop until the sum is computed for the first 32 threads in the warp
    if (tid < 32)
    {
        float sum = warpReduceSum(sharedLoss[tid]);
        if (tid == 0)
	    atomicAdd(devLoss, (float)sum);
    }
}

///////////////// FWD pass launcher
int Weighted3DTransformLoss_ForwardLauncher(const float *points, const float *masks, const float *tfms, const float *targetpts,
								  int batchSize, int ndim, int nrows, int ncols, int nSE3, int nTfmParams,
								  const long *ps, const long *ms, const long *ts,
								  hipStream_t stream)
{
    // Copy transforms to constant memory to reduce global memory read overhead
    hipMemcpyToSymbol(HIP_SYMBOL(constTfms), tfms, nTfmParams * sizeof(float));

    // Block and thread structure - we have one large set of points, so use 1d block/threads
    int npoints = batchSize * nrows * ncols;
    int numBlocks = ceil(npoints * (1.0/256));
    dim3 blocks(numBlocks);
    dim3 threads(256);

    // Allocate memory for loss on gpu
    float loss;
    float *devloss;
    hipMalloc((void**)&devloss, sizeof(float));
    hipMemset(devloss, 0, sizeof(float));

//    // Timer
//    cudaEvent_t start, stop;
//    cudaEventCreate(&start);
//    cudaEventCreate(&stop);
//    cudaEventRecord(start);

    // Project the points and run the depth test first (parallelize across number of points)
    computeLoss <<< blocks, threads, 256*sizeof(float), stream >>>(
                                                                     points,
                                                                     masks,
                                                                     targetpts,
                                                                     devloss,
                                                                     nrows,
                                                                     ncols,
                                                                     npoints,
                                                                     nSE3,
                                                                     (int) ps[0],
                                                                     (int) ps[1],
                                                                     (int) ps[2],
                                                                     (int) ps[3],
                                                                     (int) ms[0],
                                                                     (int) ms[1],
                                                                     (int) ms[2],
                                                                     (int) ms[3],
                                                                     (int) ts[0],
                                                                     (int) ts[1],
                                                                     (int) ts[2],
                                                                     (int) ts[3]
                                                                                 );

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy over the loss value
    hipMemcpy(&loss, devloss, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(devloss); // Free memory

//    // Finish timing and show stats
//    cudaEventRecord(stop);
//    cudaEventSynchronize(stop);
//    float milliseconds = 0;
//    cudaEventElapsedTime(&milliseconds, start, stop);
//    printf("FWD: Time taken in milliseconds: %f\n",milliseconds);

    // check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in Weighted3DTransformLoss_ForwardLauncher: %s\n", hipGetErrorString(err));
        assert(false);
    }

    return loss;
}

// ============= BWD PASS =================== //

// Compute the gradients w.r.t input points & masks given gradients w.r.t output 3D points
__global__ void computeLossGradients(const float *inputpts, const float *masks,
                                     float *gradInputpts, float *gradMasks, float *gradTfms,
                                     const float *targetpts, int useMaskGradMag,
                                     int nrows, int ncols, int nSE3,
                                     int ps0, int ps1, int ps2, int ps3,
                                     int ms0, int ms1, int ms2, int ms3,
                                     int ts0, int ts1, int ts2, int ts3)
{
    // Get the row, col, batch IDs & figure out if we are within limits
    int c = (blockIdx.x * blockDim.x) + threadIdx.x; // col ID (innermost dimension in our data for coalescing)
    int r = (blockIdx.y * blockDim.y) + threadIdx.y; // row ID
    int b = blockIdx.z; // Batch ID (since blockDim.z = 1, theadIdx.z = 0)
    bool withinLimits = ((c < ncols) && (r < nrows));

    // Create a shared memory buffer for storing the gradients w.r.t a single transform
    extern __shared__ float sharedData[];

    // Declare temp vars
    int tid = threadIdx.x + threadIdx.y * blockDim.x; // Id of thread in local block
    int nThreads = blockDim.x * blockDim.y;
    int nThreads2 = nThreads/2;
    int nSharedGrads  = nThreads * 12;
    int nSharedGradResults = nSE3*12;
    float *sharedGradTfms = sharedData; // nThreads*12
    float *sharedGradTfmResults = (float *)&sharedData[nSharedGrads]; // nSE3*12

    // Get 3D input point (p) & target point (gpt). Read only if inside limits
    float x, y, z, xt, yt, zt;
    int valp = b*ps0 + r*ps2 + c*ps3; // Don't add stride along 3D dim
    if (withinLimits)
    {
        x = *(inputpts + 0*ps1 + valp);
        y = *(inputpts + 1*ps1 + valp);
        z = *(inputpts + 2*ps1 + valp);

        // Get gradient w.r.t output point (gpt)
        xt = *(targetpts + 0*ps1 + valp);
        yt = *(targetpts + 1*ps1 + valp);
        zt = *(targetpts + 2*ps1 + valp);
    }

    // Compute the gradients over all the transforms from a given 3D point
    int valm = b*ms0 + r*ms2 + c*ms3;
    float gx = 0, gy = 0, gz = 0; // Grads w.r.t input pts
    for(int k = 0; k < nSE3; k++)
    {
        // Compute all the gradients if within limits or set the grads to zero
        if(withinLimits)
        {
            // Get transform & wt
            float w_k = *(masks + k*ms1 + valm);   // Get the weight for the 'k'th transform "
            float *T  = constTfms + b*ts0 + k*ts1; // Get the 'k'th transform

            // Compute transformed 3D point: p' = (R_k*p + t_k) (for X,Y,Z coordinates)
            float xp = (T[0] * x + T[1] * y + T[2]  * z + T[3]);  // (R_k * p_x + t_k)
            float yp = (T[4] * x + T[5] * y + T[6]  * z + T[7]);  // (R_k * p_y + t_k)
            float zp = (T[8] * x + T[9] * y + T[10] * z + T[11]); // (R_k * p_z + t_k)

            // Compute difference between pred & target
            float xd = (xp - xt);
            float yd = (yp - yt);
            float zd = (zp - zt);

            // === Gradient w.r.t input point (p = R^T * gpt, summed across all the "k" transforms)
            gx += w_k * (T[0] * xd + T[4] * yd + T[8]  * zd);
            gy += w_k * (T[1] * xd + T[5] * yd + T[9]  * zd);
            gz += w_k * (T[2] * xd + T[6] * yd + T[10] * zd);

            // === Gradient w.r.t mask (w_k) = (R_k^T * p + t_k) * gpt
            if (useMaskGradMag)
                *(gradMasks + k*ms1 + valm) = 0.5 * ( (xp-xt)*(xp-xt) + (yp-yt)*(yp-yt) + (zp-zt)*(zp-zt) );
            else
                *(gradMasks + k*ms1 + valm) = 0.5; // sign is always +ve

            // === Gradients w.r.t transforms (t_k), stored in shared memory
            // Grads w.r.t rotation parameters (sum across all pts)
            // First nThreads params is Tfm(0,0), next is Tfm(0,1) etc for removing memory bank conflicts when reading to shared memory
            sharedGradTfms[0*nThreads+tid]  = w_k * x * xd;
            sharedGradTfms[1*nThreads+tid]  = w_k * y * xd;
            sharedGradTfms[2*nThreads+tid]  = w_k * z * xd;
            sharedGradTfms[4*nThreads+tid]  = w_k * x * yd;
            sharedGradTfms[5*nThreads+tid]  = w_k * y * yd;
            sharedGradTfms[6*nThreads+tid]  = w_k * z * yd;
            sharedGradTfms[8*nThreads+tid]  = w_k * x * zd;
            sharedGradTfms[9*nThreads+tid]  = w_k * y * zd;
            sharedGradTfms[10*nThreads+tid] = w_k * z * zd;

            // Grads w.r.t translation parameters (sum across all pts)
            sharedGradTfms[3*nThreads+tid]  = w_k * xd;
            sharedGradTfms[7*nThreads+tid]  = w_k * yd;
            sharedGradTfms[11*nThreads+tid] = w_k * zd;
        }
        else
        {
            // Re-initialize shared memory to zero (no need to sync here as we don't += to this memory till we do a syncthreads later)
            for(int i = tid; i < nSharedGrads; i+=nThreads)
                sharedGradTfms[i] = 0;
        }
        __syncthreads(); // Synchronize all threads before we sum up the tfm gradients

        // === Do the parallel reduce for that particular transform dimension
        // === ASSUMPTION: We have power of 2 block sizes!
        // From: Slide 22 of http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
        // We use first half of threads to compute sums for first 6 transform params & the rest for the last 6 params
        for(unsigned int s = nThreads2; s>=32; s>>=1)
        {
            // Second nThreads/2 elements will be added to first nThreads/2 elements, then
            // Second nThreads/4 elements will be added to first nThreads/4 elements and so on!
            if (tid < s)
            {
                // Sum up gradients w.r.t first 6 parameters!
                for(int i = 0; i < 6; i++)
                    sharedGradTfms[i*nThreads + tid] += sharedGradTfms[i*nThreads + tid + s];
            }
            else if((tid >= nThreads2) && (tid - nThreads2) < s) // Use the second half of threads to process the remaining 6 transform parameters
            {
                // Sum up gradients w.r.t last 6 parameters!
                for(int i = 6; i < 12; i++)
                    sharedGradTfms[i*nThreads + tid - nThreads2] += sharedGradTfms[i*nThreads + tid - nThreads2 + s];
            }
            __syncthreads();
        }

        // This uses warp-shuffle to compute the sum across a warp (32 threads)
        // Note that for this to work, you have to have run the loop until the sum is computed for the first 32 threads in the warp
        if (tid < 32)
        {
            for(int i = 0; i < 12; i++)
            {
                float sum = warpReduceSum(sharedGradTfms[i*nThreads + tid]); // Declared elsewhere
                if (tid == 0)
                    sharedGradTfmResults[k*12+i] = sum; // Store final summed result in shared memory, we can copy to global later in parallel
            }
        }
    }
    __syncthreads(); // Wait till all gradients have been propely summed up!

    // Add computed tfm gradients to global memory in parallel!
    for(int i = tid; i < nSharedGradResults; i+=nThreads)
        atomicAdd(gradTfms + b*ts0 + i, sharedGradTfmResults[i]); // Final value corresponding to that term of the tfm

    // Gradients w.r.t pts (copy after sum across tfms)
    if (withinLimits)
    {
        *(gradInputpts + 0*ps1 + valp) = gx;
        *(gradInputpts + 1*ps1 + valp) = gy;
        *(gradInputpts + 2*ps1 + valp) = gz;
    }
}

////////////////////////////////////
// == BWD pass code
void Weighted3DTransformLoss_BackwardLauncher(const float *points, const float *masks, const float *tfms, const float *targetpts,
                                              float *gradPoints, float *gradMasks, float *gradTfms, int useMaskGradMag,
                                              int batchSize, int ndim, int nrows, int ncols, int nSE3, int nTfmParams,
                                              const long *ps, const long *ms, const long *ts,
                                              hipStream_t stream)
{
    // Copy transforms to constant memory to reduce global memory read overhead
    hipMemcpyToSymbol(HIP_SYMBOL(constTfms), tfms, nTfmParams * sizeof(float));

    // Compute gradients w.r.t the input tfms next
    dim3 threads(16,16,1);
    dim3 blocks(ceil(ncols*(1.0/threads.x)),ceil(nrows*(1.0/threads.y)),batchSize); // all threads in a block will access same example
    int sharedMemSize = threads.x * threads.y * 3 * 4 * sizeof(float) + nSE3 * 3 * 4 * sizeof(float); // Memory for 12 vals per thread + nSE3*12 vals for storing result
    if (sharedMemSize > 32000)
    {
        printf("Shared memory size for transform gradients (%d) > 32000. Can't be stored in shared memory."
               "Please use NonRigidTransform3D layer + MSE criterion or reduce number of threads per block \n", sharedMemSize);
        assert(false); // Exit
    }

//    // Timer
//    cudaEvent_t start, stop;
//    cudaEventCreate(&start);
//    cudaEventCreate(&stop);
//    cudaEventRecord(start);

    computeLossGradients<<< blocks, threads, sharedMemSize, stream >>>(
                                                                        points,
                                                                        masks,
                                                                        gradPoints,
                                                                        gradMasks,
                                                                        gradTfms,
                                                                        targetpts,
                                                                        useMaskGradMag,
                                                                        nrows,
                                                                        ncols,
                                                                        nSE3,
                                                                        (int) ps[0],
                                                                        (int) ps[1],
                                                                        (int) ps[2],
                                                                        (int) ps[3],
                                                                        (int) ms[0],
                                                                        (int) ms[1],
                                                                        (int) ms[2],
                                                                        (int) ms[3],
                                                                        (int) ts[0],
                                                                        (int) ts[1],
                                                                        (int) ts[2],
                                                                        (int) ts[3]
                                                                                    );

    // Wait for kernel to finish
    hipDeviceSynchronize();

//    // Finish timing and show stats
//    cudaEventRecord(stop);
//    cudaEventSynchronize(stop);
//    float milliseconds = 0;
//    cudaEventElapsedTime(&milliseconds, start, stop);
//    printf("BWD: Time taken in milliseconds: %f\n",milliseconds);

    // check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in Weighted3DTransformLoss_BackwardLauncher: %s\n", hipGetErrorString(err));
        assert(false);
    }
}

#ifdef __cplusplus
}
#endif
